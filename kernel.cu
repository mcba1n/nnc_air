#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>

#include <stdio.h>
#include <iostream>
#include <random>
#include <fstream>
#include <string>
#include <sstream>

#include "csv.h"

#define SQRT1_2   0.707106781
#define SQRT1_2PI 0.398942280

// GFA
__host__ double gfa_run(int m, int Ns, int Nd, int max_int_len, double* y, double* P, double* P_K, int* intervals, double sigma, double* f);
__global__ void gfa_slice(int ell, int Ns, int Nd, int max_int_len, double* y, double* Fp, double* Fc,
    double* P, double* P_K, int* intervals, double sigma, double* f);
__global__ void gfa_init(int Ns, int Nd, int max_int_len, double* y, double* F,
    double* P, double* P_K, double sigma, double* f, int s_0);
__global__ void gfa_termination(int m, int Ns, int max_int_len, double* F, int* intervals, double* log_post_vec);

// GCA
__host__ double gca_run(int m, int* s_seq, int Nd, int max_int_len, double* y,
    double* P_K, int* intervals, double sigma, double* f);
__global__ void gca_slice(int ell, int s, int Nd, int max_int_len, double* y,
    double* Cp, double* Cc, double* P_K, int* intervals, double sigma, double* f);
__global__ void gca_init(int Nd, int max_int_len, double* y, double* C,
    double* P_K, double sigma, double* f, int s, int s_0);
__global__ void gca_termination(int m, int max_int_len, double* C, int* intervals, double* log_post_vec);

// Helpers
__host__ __device__ double eln(double x);
__host__ __device__ double eexp(double x);
__host__ __device__ double elnproduct(double x, double y);
__host__ __device__ double elnsum(double x, double y);

__host__ __device__ double normalPDF(double value, double mu, double sigma);
__host__ thrust::host_vector<int> bd_int(int m, int W, int T, double E_K);
std::vector<std::vector<double>> read_data(std::string fname, bool f);


int main()
{
    //printf("result: %.4f\n", elnsum(eln(0), -60));

    std::cout << "air_sim3" << std::endl;
    std::default_random_engine generator;
    //std::srand(std::time(NULL));

    std::ofstream datafile;
    datafile.open("model_biawgn/data.csv");
    //data_genie_int_rad0

    // Markov probabilities
    std::vector<std::vector<double>> P = read_data("model_biawgn/P.csv", 0);
    int Ns = P[0].size();
    thrust::host_vector<double> Ph(Ns * Ns);
    for (int i = 0; i < Ns; i++) for (int j = 0; j < Ns; j++) Ph[i * Ns + j] = P[i][j];

    // Channel mapping
    std::vector<std::vector<double>> f = read_data("model_biawgn/f.csv", 0);
    thrust::host_vector<double> fh(Ns);
    for (int i = 0; i < Ns; i++) fh[i] = f[i][0];

    // Duration probabilities
    std::vector<std::vector<double>> P_K = read_data("model_biawgn/P_K.csv", 0);
    int k_max = P_K.size();
    thrust::host_vector<double> P_Kh(k_max);
    for (int i = 0; i < k_max; i++) P_Kh[i] = P_K[i][0] + std::numeric_limits<double>::epsilon();
    double E_K = 0;
    for (int k = 0; k < k_max; k++) E_K += (k + 1) * P_Kh[k];
    int W = 0;
    for (int i = 0; i < k_max; i++) if (P_Kh[i] > std::numeric_limits<double>::epsilon()) W += 1;

    // Run simulations
    const double sigma_min = std::numeric_limits<double>::epsilon();
    const double sigma_max = 1.0;
    const double delta = 0.05;
    const int N_dps = round((sigma_max - sigma_min) / delta) + 1;

    std::vector<double> sigma_vals(N_dps);
    std::vector<double> air_vals(N_dps);

    for (int sim_idx = 0; sim_idx < N_dps; sim_idx++) {
        // Generator random signal
        double sigma = sigma_min + sim_idx * delta;
        const int m = 100000;
        const int s_0 = 0;
        thrust::host_vector<double> y;
        thrust::host_vector<int> s_seq(m);
        thrust::host_vector<int> t_seq(m);
        int s_prev = s_0;

        for (int i = 0; i < m; i++) {
            int k = std::discrete_distribution<uint64_t>(P_Kh.begin(), P_Kh.end())(generator) + 1;
            int s = std::discrete_distribution<uint64_t>(P[s_prev].begin(), P[s_prev].end())(generator);
            std::normal_distribution<double> norm_dist(fh[s], sigma);
            for (int j = 0; j < k; j++) y.push_back(norm_dist(generator));
            s_seq[i] = s;
            s_prev = s;
            if (i == 0) t_seq[0] = k;
            else t_seq[i] = t_seq[i - 1] + k;
        }

        // Intervals
        int T = y.size();
        thrust::host_vector<int> intervals(m * 2);
        intervals = bd_int(m, W, T, E_K);
        //intervals = bd_int_markers(m, T, t_seq);
        std::vector<int> int_lens(m);
        for (int i = 0; i < m; i++) int_lens[i] = intervals[i * 2 + 1] - intervals[i * 2] + 1;
        int max_int_len = *max_element(int_lens.begin(), int_lens.end());

         
        // Convert host vectors to device vectors
        thrust::device_vector<double> Pd(Ns * Ns);
        Pd = Ph;
        thrust::device_vector<double> fd(Ns);
        fd = fh;
        thrust::device_vector<double> P_Kd(k_max);
        P_Kd = P_Kh;
        thrust::device_vector<int> intervalsd(m * 2);
        intervalsd = intervals;
        thrust::device_vector<double> yd;
        yd = y;
        thrust::device_vector<int> s_seqd(m);
        s_seqd = s_seq;

        // Forward probabilities
        double log_post = gfa_run(m, Ns, k_max, max_int_len, thrust::raw_pointer_cast(yd.data()), thrust::raw_pointer_cast(Pd.data()),
            thrust::raw_pointer_cast(P_Kd.data()), thrust::raw_pointer_cast(intervalsd.data()), sigma, thrust::raw_pointer_cast(fd.data()));
        //printf("H_Y: %.4f\n", -log_post/(log(2)*m));
        
        // Conditional probailities
        double log_post_cond = gca_run(m, thrust::raw_pointer_cast(s_seq.data()), k_max, max_int_len, thrust::raw_pointer_cast(yd.data()),
            thrust::raw_pointer_cast(P_Kd.data()), thrust::raw_pointer_cast(intervalsd.data()), sigma, thrust::raw_pointer_cast(fd.data()));
        //printf("H_YS: %.4f\n", -log_post_cond / (log(2) * m));

        // Information rates
        double H_Y = -log_post / (log(2) * m);
        double H_Y_S = -log_post_cond / (log(2) * m);
        double I_nnc = H_Y - H_Y_S;

        // Record data
        air_vals[sim_idx] = I_nnc;
        sigma_vals[sim_idx] = sigma;
        datafile << std::to_string(sigma) << "," << std::to_string(I_nnc) << "\n";
        printf("sigma: %.4f, I: %.4f\n", sigma, I_nnc);
    }

    return 0;
}


// GFA:

__global__ void gfa_init(int Ns, int Nd, int max_int_len, double* y, double* F,
    double* P, double* P_K, double sigma, double* f, int s_0) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int s = idx / max_int_len;
    int t_shifted = idx % max_int_len;

    double log_gamma = elnproduct(eln(P[s_0 * Ns + s]), eln(P_K[t_shifted]));
    for (int j = 0; j < t_shifted; j++) log_gamma = elnproduct(log_gamma, eln(normalPDF(y[j], f[s], sigma)));

    int F_idx = (s * max_int_len) + t_shifted;
    F[F_idx] = log_gamma;
}

__global__ void gfa_termination(int m, int Ns, int max_int_len, double* F, int* intervals, double* log_post_vec) {
    // does termination after ell=m
    int last_shifted_idx = intervals[(m - 1) * 2 + 1] - intervals[(m - 1) * 2];
    for (int s = 0; s < Ns; s++) {
        int F_idx = (s * max_int_len) + last_shifted_idx;
        log_post_vec[0] = elnsum(log_post_vec[0], F[F_idx]);
    }
}

__host__ double gfa_run(int m, int Ns, int Nd, int max_int_len, double* y, double* P, double* P_K, int* intervals, double sigma, double* f) {
    //printf("Running GFA...\n");
    hipError_t cudaStatus;
    int N = max_int_len * Ns;
    int THREADS_PER_BLOCK = 800;
    int NUM_BLOCKS = N / THREADS_PER_BLOCK + 1;

    thrust::host_vector<double> Fhp(Ns * max_int_len, eln(0));
    thrust::host_vector<double> Fhc(Ns * max_int_len, eln(0));
    thrust::device_vector<double> Fdp(Ns * max_int_len, eln(0));
    thrust::device_vector<double> Fdc(Ns * max_int_len, eln(0));
    
    // Init
    gfa_init<<<NUM_BLOCKS, THREADS_PER_BLOCK >>>(Ns, Nd, max_int_len, y, thrust::raw_pointer_cast(Fdp.data()), P, P_K, sigma, f, 0);
    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    
    // Slices
    for (int ell = 2; ell <= m; ell++) { 
        gfa_slice << <NUM_BLOCKS, THREADS_PER_BLOCK>> > (ell, Ns, Nd, max_int_len, y, thrust::raw_pointer_cast(Fdp.data()), 
            thrust::raw_pointer_cast(Fdc.data()), P, P_K, intervals, sigma, f);
        hipDeviceSynchronize();
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        
        /*
        if (ell == m) {
            printf("DATA1:\n");
            Fhc = Fdc;
            for (int t = 0; t < max_int_len; t++) {
                for (int s = 0; s < Ns; s++) {
                    int F_idx = (s * max_int_len) + t;
                    printf("(s: %d, t: %d): %.4f\n", s, t, Fhc[F_idx]);
                }
            }
            printf("DATA2:\n");
            Fhp = Fdp;
            for (int t = 0; t < max_int_len; t++) {
                for (int s = 0; s < Ns; s++) {
                    int F_idx = (s * max_int_len) + t;
                    printf("(s: %d, t: %d): %.4f\n", s, t, Fhp[F_idx]);
                }
            }
        }*/

        // Shift slices in F
        thrust::copy(thrust::device, Fdc.begin(), Fdc.end(), Fdp.begin());
        thrust::fill(thrust::device, Fdc.begin(), Fdc.end(), eln(0));

        printf("GFA, ell = %d\n", ell);
    }

    // Termination
    thrust::device_vector<double> log_post_d(1, eln(0));
    thrust::host_vector<double> log_post_h(1, eln(0));
    gfa_termination<<<1,1>>>(m, Ns, max_int_len, thrust::raw_pointer_cast(Fdp.data()), intervals, thrust::raw_pointer_cast(log_post_d.data()));
    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    log_post_h = log_post_d;
    return log_post_h[0];
}


__global__ void gfa_slice(int ell, int Ns, int Nd, int max_int_len, double* y, double* Fp, double* Fc, 
    double* P, double* P_K, int* intervals, double sigma, double* f) {
    //NOTE: t should be in I_{ell}, t_prev should be in I_{ell-1}
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int s = idx / max_int_len;
    int t_shifted = idx % max_int_len;
    int t = intervals[(ell - 1) * 2] + t_shifted; 

    int max_int_len_ell_curr = intervals[(ell - 1) * 2 + 1] - intervals[(ell - 1) * 2] + 1;
    int max_int_len_ell_prev = intervals[(ell - 2) * 2 + 1] - intervals[(ell - 2) * 2] + 1;
    if (t_shifted > max_int_len_ell_curr - 1) return; // GPU should not run this kernel if t is outside of its interval

    for (int s_prev = 0; s_prev < Ns; s_prev++) {
        if (P[s_prev * Ns + s] == 0.0) continue;
        for (int k = 0; k < Nd; k++) {
            int t_start = t - k;
            int t_prev = t_start - 1;

            int t_prev_shifted = t_prev - intervals[(ell - 2) * 2]; // the segment starts here, 1 sample after t_prev; this must be in the interval
            if (t_prev_shifted < 0 || t_prev_shifted > max_int_len_ell_prev - 1) continue; // skip t_prev if it is not inside its interval

            double log_gamma = elnproduct(eln(P[s_prev * Ns + s]), eln(P_K[k]));
            for (int tt = t_start; tt <= t; tt++) log_gamma = elnproduct(log_gamma, eln(normalPDF(y[tt], f[s], sigma)));

            int F_prev_idx = (s_prev * max_int_len) + t_prev_shifted;
            int F_idx = (s * max_int_len) + t_shifted;
            Fc[F_idx] = elnsum(Fc[F_idx], elnproduct(log_gamma, Fp[F_prev_idx]));
        }
    }
}


// GCA:
__global__ void gca_init(int Nd, int max_int_len, double* y, double* C,
    double* P_K, double sigma, double* f, int s, int s_0) {
    int t_shifted = blockIdx.x * blockDim.x + threadIdx.x;

    double log_gamma = eln(P_K[t_shifted]);
    for (int j = 0; j < t_shifted; j++) log_gamma = elnproduct(log_gamma, eln(normalPDF(y[j], f[s], sigma)));

    int C_idx = t_shifted;
    C[C_idx] = log_gamma;
}

__global__ void gca_termination(int m, int max_int_len, double* C, int* intervals, double* log_post_vec) {
    int last_shifted_idx = intervals[(m - 1) * 2 + 1] - intervals[(m - 1) * 2];
    int C_idx = last_shifted_idx;
    log_post_vec[0] = elnsum(log_post_vec[0], C[C_idx]);
}

__host__ double gca_run(int m, int* s_seq, int Nd, int max_int_len, double* y,
    double* P_K, int* intervals, double sigma, double* f) {
    //printf("Running GCA...\n");
    hipError_t cudaStatus;
    int N = max_int_len;
    int THREADS_PER_BLOCK = 800;
    int NUM_BLOCKS = N / THREADS_PER_BLOCK + 1;

    thrust::host_vector<double> Fhp(max_int_len, eln(0));
    thrust::host_vector<double> Fhc(max_int_len, eln(0));
    thrust::device_vector<double> Fdp(max_int_len, eln(0));
    thrust::device_vector<double> Fdc(max_int_len, eln(0));

    // Init
    gca_init << <NUM_BLOCKS, THREADS_PER_BLOCK >> > (Nd, max_int_len, y, thrust::raw_pointer_cast(Fdp.data()), P_K, sigma, f, s_seq[0], 0);
    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    
    // Slices
    for (int ell = 2; ell <= m; ell++) {
        gca_slice << <NUM_BLOCKS, THREADS_PER_BLOCK >> > (ell, s_seq[ell-1], Nd, max_int_len, y, thrust::raw_pointer_cast(Fdp.data()),
            thrust::raw_pointer_cast(Fdc.data()), P_K, intervals, sigma, f);
        hipDeviceSynchronize();
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    
        // Shift slices in F
        thrust::copy(thrust::device, Fdc.begin(), Fdc.end(), Fdp.begin());
        thrust::fill(thrust::device, Fdc.begin(), Fdc.end(), eln(0));

        printf("GCA, ell = %d\n", ell);
    }
    
    // Termination
    thrust::device_vector<double> log_post_d(1, eln(0));
    thrust::host_vector<double> log_post_h(1, eln(0));
    gca_termination << <1, 1 >> > (m, max_int_len, thrust::raw_pointer_cast(Fdp.data()), intervals, thrust::raw_pointer_cast(log_post_d.data()));
    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    log_post_h = log_post_d;
    return log_post_h[0];
}


__global__ void gca_slice(int ell, int s, int Nd, int max_int_len, double* y,
    double* Cp, double* Cc, double* P_K, int* intervals, double sigma, double* f) {
    int t_shifted = blockIdx.x * blockDim.x + threadIdx.x;
    int t = intervals[(ell - 1) * 2] + t_shifted;

    int max_int_len_ell_curr = intervals[(ell - 1) * 2 + 1] - intervals[(ell - 1) * 2] + 1;
    int max_int_len_ell_prev = intervals[(ell - 2) * 2 + 1] - intervals[(ell - 2) * 2] + 1;
    if (t_shifted > max_int_len_ell_curr - 1) return;

    for (int k = 0; k < Nd; k++) {
        int t_start = t - k;
        int t_prev = t_start - 1;

        int t_prev_shifted = t_prev - intervals[(ell - 2) * 2];
        if (t_prev_shifted < 0 || t_prev_shifted > max_int_len_ell_prev - 1) continue;

        double log_gamma = eln(P_K[k]);
        for (int tt = t_start; tt <= t; tt++) log_gamma = elnproduct(log_gamma, eln(normalPDF(y[tt], f[s], sigma)));

        int C_prev_idx = t_prev_shifted;
        int C_idx = t_shifted;
        Cc[C_idx] = elnsum(Cc[C_idx], elnproduct(log_gamma, Cp[C_prev_idx]));
    }
}



// HELPER FUNCTIONS:

__host__ __device__ double elnsum(double x, double y) {
    double z;
    if (x == -INFINITY || y == -INFINITY) {
        if (x == -INFINITY) z = y;
        else z = x;
    }
    else {
        if (x > y) z = x + log1p(exp(y - x));
        else z = y + log1p(exp(x - y));
    }
    return z;
}

__host__ __device__ double normalPDF(double value, double mu, double sigma) {
    return 1 / sigma * SQRT1_2PI * exp(-pow(value - mu, 2) / (2 * pow(sigma, 2)));
}

__host__ thrust::host_vector<int> bd_int(int m, int W, int T, double E_K) {
    thrust::host_vector<int> intervals(m * 2);
    double eps = 1e-10;
    double rho = W * log(2 / eps) / 2;
    for (int ell = 1; ell <= m; ell++) {
        int t = ceil(sqrt(ell * rho));
        int i_lb = floor((ell - 1) * E_K - t);
        int i_ub = ceil(ell * E_K + t);

        intervals[(ell - 1) * 2] = std::max(1, i_lb) - 1;
        intervals[(ell - 1) * 2 + 1] = std::min(T, i_ub) - 1;
    }
    return intervals;
}


__host__ __device__ double eln(double x) {
    if (x == 0) return -INFINITY;
    else return log(x);
}

__host__ __device__ double eexp(double x) {
    if (x == -INFINITY)  return 0;
    else return exp(x);
}

__host__ __device__ double elnproduct(double x, double y) {
    if (x == -INFINITY || y == -INFINITY) return -INFINITY;
    else return x + y;
}

std::vector<std::vector<double>> read_data(std::string fname, bool f = 0) {
    std::vector<std::vector<double>> data;
    std::ifstream inFile;
    // open the file stream
    inFile.open(fname);
    // check if opening a file failed
    if (inFile.fail()) {
        std::cerr << "Error opening file" << std::endl;
        inFile.close();
        exit(1);
    }
    std::string line;
    while (getline(inFile, line))
    {
        std::vector<double> vect;
        std::stringstream ss(line);
        for (double i; ss >> i;) {
            vect.push_back(i);
            if (ss.peek() == ',')
                ss.ignore();
        }
        data.push_back(vect);

    }
    // close the file stream
    inFile.close();

    // print if flag is set
    if (f) {
        for (int i = 0; i < data.size(); i++) {
            for (int j = 0; j < data[0].size(); j++) {
                std::cout << "(" << i << ", " << j << "): " << data[i][j] << std::endl;
            }
        }
    }

    return data;
}